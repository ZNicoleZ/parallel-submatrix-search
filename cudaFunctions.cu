#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#define BLOCKS 8
#define THREADS 1024

__device__ double calcDiff(int p, int o) {

	// utility function for calculating difference between Picture element and an Object element
	
	double dif = p-o;
	dif = abs(dif);
	dif = dif/p;
	return dif;
}

__device__ double calcMatrixMatch(int* picMat, int picDim, int* objMat, int objDim, int i, int j) {

	// utility function for calculating the overall matching value of a picture and an object in position i,j
	
	double match_sum = 0;
	for(int r = 0; r<objDim; r++){
		for(int c=0; c<objDim; c++){
			if(r+i < picDim && c+j < picDim)
				match_sum += calcDiff(picMat[(i+r)*picDim + j+c], objMat[r*objDim + c]);
			else
				return -1;
		}
	}

	return match_sum/(objDim*objDim);
}


__global__ void kernelCheckMatrixRangeMatch(int *d_pic, int picDim, int *d_obj, int objDim, double matchValue, int* d_ind, int fromRow , int toRow, int fromCol, int toCol) {
	
	int i = blockIdx.x * blockDim.x + threadIdx.x;	// set a "global" thread index for each thread
    	int numRows = toRow - fromRow;			// set row range
	int numCols = toCol - fromCol;			// set column range
	
	if(numRows <= 0 || numCols <= 0) {
    		return;
    	}
    	
    	int row = (i)/(numRows) + fromRow;		// define position's row to check
	int col = (i)%(numRows) + fromCol;		// define position's column to check

	if (row >= fromRow && row <= toRow && col >= fromCol && col <= toCol) {	// if the position is in the given range, calculate matching value
		
		double match = calcMatrixMatch(d_pic, picDim, d_obj, objDim, row, col);
		
		if(match<=matchValue && match!=-1){	// if object was found, set the found index in the result pointer (d_ind)
			*d_ind = (row*picDim + col);
			return;
		}
	}

}

int copyDataToGPU(int* picMat, int picDim, int* objMat, int objDim, int** dev_pic, int** dev_obj, int** dev_ind) {

	hipError_t err = hipSuccess;

	size_t picSize = picDim * picDim * sizeof(int);
	size_t objSize = objDim * objDim * sizeof(int);

	// Allocate memory on GPU to copy the data from the host
	int *d_pic;
	err = hipMalloc((void **)&d_pic, picSize);
	if (err != hipSuccess) {
		fprintf(stderr, "Failed to allocate device memory (picture) - %s\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	int *d_obj;
	err = hipMalloc((void **)&d_obj, objSize);
	if (err != hipSuccess) {
		fprintf(stderr, "Failed to allocate device memory (object) - %s\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	
	int h_ind = -1;
	int* d_ind;
	err = hipMalloc((void**)&d_ind, sizeof(int));
	if (err != hipSuccess) {
		fprintf(stderr, "Failed to allocate device memory (index) - %s\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	// Copy data from host to the GPU memory
	err = hipMemcpy(d_pic, picMat, picSize, hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		fprintf(stderr, "Failed to copy data from host to device (picture) - %s\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	err = hipMemcpy(d_obj, objMat, objSize, hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		fprintf(stderr, "Failed to copy data from host to device (object) - %s\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
    
	err = hipMemcpy(d_ind, &h_ind, sizeof(int), hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		fprintf(stderr, "Failed to copy data from host to device (index) - %s\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	
	*dev_pic = d_pic;
	*dev_obj = d_obj;
	*dev_ind = d_ind;
	
	return 0;
}

int compareThreadMatrixOnGPU(int *d_pic, int picDim, int *d_obj, int objDim, double matchValue, int* d_ind, int fromRow , int toRow, int fromCol, int toCol) {

	hipError_t err = hipSuccess;
	int h_ind;
	
	int rows = toRow - fromRow;
	int cols = toCol - fromCol;
	int size = (rows + 1) * (cols + 1);
	int blocks = (size + THREADS - 1) / THREADS;	

	// Launch the Kernel
	kernelCheckMatrixRangeMatch<<<blocks, THREADS>>>(d_pic, picDim, d_obj, objDim, matchValue, d_ind, fromRow , toRow, fromCol, toCol);
	err = hipGetLastError();

	if (err != hipSuccess) {
		fprintf(stderr, "Failed to launch 'kernelCheckMatrixRangeMatch()' -  %s\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	// Copy the  result from GPU to the host memory.
	err = hipMemcpy(&h_ind, d_ind, sizeof(int), hipMemcpyDeviceToHost);
	if (err != hipSuccess) {
		fprintf(stderr, "Failed to copy data from device to host - %s\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	
	return h_ind;
	
}


int freeCudaData(int *d_pic, int *d_obj, int* d_ind) {

	hipError_t err = hipSuccess;
	
	// Free allocated memory on GPU
	if (hipFree(d_pic) != hipSuccess) {
		fprintf(stderr, "Failed to free device data (picture) - %s\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	if (hipFree(d_obj) != hipSuccess) {
		fprintf(stderr, "Failed to free device data (object) - %s\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	
	if (hipFree(d_ind) != hipSuccess) {
		fprintf(stderr, "Failed to free device data (index) - %s\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	
	return 0;

}


